#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018-2019 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.

  Additional permission under GNU GPL version 3 section 7

  If you modify this Program, or any covered work, by linking or
  combining it with NVIDIA Corporation's libraries from the NVIDIA CUDA
  Toolkit and the NVIDIA CUDA Deep Neural Network library (or a
  modified version of those libraries), containing parts covered by the
  terms of the respective license agreement, the licensors of this
  Program grant you additional permission to convey the resulting work.
*/

#include "cuda_common.h"
#include "neural/network.h"

namespace lczero {
namespace cudnn_backend {

/////////////////////////////////////////////////////////////////////////////
//          Simple CUDA kernels used by certain layers                     //
/////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void addVectors_kernel(T* c, T* a, T* b, int size, int asize,
                                  int bsize, bool relu, bool useTanh,
                                  bool useSigmoid) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size) {
    float aVal = 0;
    float bVal = 0;
    if (a) aVal = (float)(a[i % asize]);
    if (b) bVal = (float)(b[i % bsize]);

    float cVal = aVal + bVal;

    if (relu && (cVal < 0)) cVal = 0;

    if (useTanh) {
      cVal = tanh(cVal);
    }

    if (useSigmoid) {
      cVal = 1.0f / (1.0f + exp(-cVal));
    }

    c[i] = (T)cVal;
  }
}

// Adds two vectors (possibly of different sizes), also do optional relu
// activation.
template <typename T>
void addVectors(T* c, T* a, T* b, int size, int asize, int bsize, bool relu,
                bool use_tanh, bool use_sigmoid) {
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addVectors_kernel<<<blocks, kBlockSize>>>(c, a, b, size, asize, bsize, relu,
                                            use_tanh, use_sigmoid);
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void addBias_NCHW_kernel(T* c, T* a, T* b, int N, int C, int H,
                                    int W) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int size = N * C * H * W;
  if (i < size) {
    float aVal = (float)a[i];

    // All this math can be optimized, but the kernel is memory bound anyway.
    int biasIndex = (i / (H * W)) % C;
    float bVal = (float)b[biasIndex];

    float cVal = aVal + bVal;
    c[i] = (T)cVal;
  }
}

// Add bias to convolution's output.
template <typename T>
void addBias_NCHW(T* c, T* a, T* b, int N, int C, int H, int W) {
  int size = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(size, kBlockSize);

  addBias_NCHW_kernel<<<blocks, kBlockSize>>>(c, a, b, N, C, H, W);
  ReportCUDAErrors(hipGetLastError());
}

__device__ half readNCHW(float* input_tensor, int n, int c, int h, int w,
                         int Nin, int Cin, int H, int W) {
  if (n >= Nin || c >= Cin) return 0;

  int index;
  index = n;
  index *= Cin;
  index += c;
  index *= H;
  index += h;
  index *= W;
  index += w;

  return (half)(input_tensor[index]);
}

__global__ void fp32NCHWtofp16NHWC_kernel(half* output_tensor,
                                          float* input_tensor, int Nin, int Cin,
                                          int Nout, int Cout, int H, int W) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= Nout * Cout * H * W) return;

  int index = tid;

  int c = (index % Cout);
  index /= Cout;
  int w = index % W;
  index /= W;
  int h = index % H;
  index /= H;
  int n = index;

  output_tensor[tid] = readNCHW(input_tensor, n, c, h, w, Nin, Cin, H, W);
}

void fp32NCHWtofp16NHWC(half* output_tensor, float* input_tensor, int Nin,
                        int Cin, int Nout, int Cout, int H, int W) {
  size_t numElements = Nout * Cout * H * W;
  const int blockSize = 256;
  int blocks = DivUp(numElements, blockSize);
  fp32NCHWtofp16NHWC_kernel<<<blocks, blockSize>>>(output_tensor, input_tensor,
                                                   Nin, Cin, Nout, Cout, H, W);
}

template <typename DstType, typename SrcType>
__global__ void copyTypeConverted_kernel(DstType* op, SrcType* ip, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= N) return;

  DstType el = (DstType)ip[tid];
  op[tid] = el;
}

template <typename DstType, typename SrcType>
void copyTypeConverted(DstType* op, SrcType* ip, int N) {
  const int kBlockSize = 256;
  int blocks = DivUp(N, kBlockSize);
  copyTypeConverted_kernel<<<blocks, kBlockSize>>>(op, ip, N);
}

template <typename T>
__global__ void batchNorm_kernel(T* output, const T* input, const T* skipInput,
                                 int N, int C, int H, int W, const float* means,
                                 const float* varMultipliers, bool relu) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int wIndex = 0;
  if (sizeof(T) == sizeof(float))
    wIndex = (index / (H * W)) % C;  // NCHW for fp32.
  else
    wIndex = index % C;  // NHWC for fp16.

  float el = input[index];
  float mean = means[wIndex];
  float varMulti = varMultipliers[wIndex];

  el -= mean;
  el *= varMulti;

  if (skipInput) el += (float)skipInput[index];

  if (relu && (el < 0)) el = 0;

  output[index] = (T)el;
}

// Every thread processes single element.
template <typename T>
void batchNorm(T* output, const T* input, const T* skipInput, int N, int C,
               int H, int W, float* means, float* var_multipliers, bool relu) {
  const int total_elements = N * C * H * W;
  const int kBlockSize = 256;
  int blocks = DivUp(total_elements, kBlockSize);

  batchNorm_kernel<<<blocks, kBlockSize>>>(output, input, skipInput, N, C, H, W,
                                           means, var_multipliers, relu);

  ReportCUDAErrors(hipGetLastError());
}

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  // Block size of 256, same mask/val for 64 consecutive threads.
  constexpr int kNumShmemElments = 256 / 64;

  __shared__ uint64_t shMasks[kNumShmemElments];
  __shared__ float shVals[kNumShmemElments];

  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int planeIndex = index >> 6;

  if (planeIndex >= n) return;

  // Load inputs to shared memory.
  if (threadIdx.x < kNumShmemElments) {
    shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
    shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
  }
  __syncthreads();

  uint64_t mask = shMasks[threadIdx.x >> 6];

  int sqIndex = index & 0x3F;
  float op = 0;

  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    op = shVals[threadIdx.x >> 6];
  }
  output[index] = op;
}

void expandPlanes_Fp32_NCHW(float* output, const uint64_t* masks,
                            const float* values, int n) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int blockSize = 256;
  int blocks = DivUp(threads, blockSize);
  expandPlanes_kernel_Fp32_NCHW<<<blocks, blockSize>>>(output, masks, values,
                                                       n);
  ReportCUDAErrors(hipGetLastError());
}

// TODO: Can optimize using shared memory if this becomes a bottleneck.
__global__ void expandPlanes_kernel_Fp16_NHWC(half* output,
                                              const uint64_t* masks,
                                              const float* values, int n) {
  const int index = threadIdx.x + blockDim.x * blockIdx.x;
  if (index >= n * 8 * 8) return;

  const int planeIndex = index % kInputPlanes;
  const int boardIndex = index / (kInputPlanes * 8 * 8);
  const int sqIndex = (index / kInputPlanes) & 0x3F;

  uint64_t mask = masks[boardIndex * kInputPlanes + planeIndex];

  half op = 0;
  bool set = !!(mask & (1ull << sqIndex));
  if (set) {
    float val = values[boardIndex * kInputPlanes + planeIndex];
    op = (half)val;
  }
  output[index] = op;
}

void expandPlanes_Fp16_NHWC(half* output, const uint64_t* masks,
                            const float* values, int n) {
  int threads = n * 8 * 8;  // Each thread writes a single element.
  const int kBlockSize = 256;
  int blocks = DivUp(threads, kBlockSize);
  expandPlanes_kernel_Fp16_NHWC<<<blocks, kBlockSize>>>(output, masks, values,
                                                        n);
  ReportCUDAErrors(hipGetLastError());
}

__global__ void globalScale_kernel(float* output, const float* input,
                                   const float* scaleBias,
                                   const float* prevLayerBias, int inputSize,
                                   int C) {
  const int kPlaneSize = 64;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid > inputSize) return;

  int nc = tid / kPlaneSize;
  int n = nc / C;
  int c = nc % C;

  float val1 = input[tid];   // Output of residual block to be scaled.
  float val2 = output[tid];  // Skip connection to be added directly.

  if (prevLayerBias) {
    val1 += prevLayerBias[c];
  }

  int startIdx = n * 2 * C;  // Scale and bias interleaved.

  float s = scaleBias[startIdx + c];
  s = 1.0f / (1.0f + exp(-s));  // Sigmoid on scale.

  float b = scaleBias[startIdx + c + C];

  float op = val1 * s + val2 + b;
  if (op < 0) op = 0;
  output[tid] = op;
}

__global__ void globalScale_kernel_fp16_nhwc(half* output, const half* input,
                                             const half* scaleBias,
                                             const half* prevLayerBias,
                                             int inputSize, int C, int HWC) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid > inputSize) return;

  int c = tid % C;
  int n = tid / (HWC);

  float val1 = (float)input[tid];   // Output of residual block to be scaled.
  float val2 = (float)output[tid];  // Skip connection to be added directly.
  if (prevLayerBias) {
    val1 += (float)prevLayerBias[c];
  }

  int startIdx = n * 2 * C;  // Scale and bias interleaved.

  float s = scaleBias[startIdx + c];
  s = 1.0f / (1.0f + exp(-s));  // Sigmoid on scale.

  float b = scaleBias[startIdx + c + C];

  float op = val1 * s + val2 + b;
  if (op < 0) op = 0;

  output[tid] = (half)op;
}

// N blocks.
// C threads per block.
// 'HWC' input data processed by thread block.
// Each thread writes a single output.
__global__ void globalAvgPool_kernel_NHWC_fp16(half* output, const half* input,
                                               const half* prevLayerBias,
                                               int inputSize, int outputSize) {
  const int elementsPerThread = 64;  // 8x8 board.

  int blockStart = blockIdx.x * blockDim.x;

  float S = 0;

#pragma unroll
  for (int i = 0; i < elementsPerThread; i++) {
    int localIndex = i * blockDim.x + threadIdx.x;
    int inputIndex = blockStart * elementsPerThread + localIndex;
    if (inputIndex < inputSize) S += (float)(input[inputIndex]);
  }

  float avg = S / elementsPerThread;

  // Add bias from previous layer.
  if (prevLayerBias) avg += (float)(prevLayerBias[threadIdx.x]);

  int opIndex = blockStart + threadIdx.x;
  if (opIndex < outputSize) output[opIndex] = (half)avg;
}

// Each thread reads 2 inputs (8x8/32), and each warp writes a single output.
__global__ void globalAvgPool_kernel(float* output, const float* input,
                                     const float* prevLayerBias, int inputSize,
                                     int outputSize, int C) {
  const int elementsPerWarp = 64;
  const int elementsPerThread = 2;

  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int laneId = threadIdx.x & 0x1F;
  int laneStartIndex = (tid - laneId) * elementsPerThread;

  // Compute per-thread sum for elementsPerThread elements.
  float S = 0;

#pragma unroll
  for (int i = 0; i < elementsPerWarp; i += 32) {
    int index = laneStartIndex + laneId + i;
    if (index < inputSize) S += input[index];
  }

// Compute warp wide sum (for entire plane - elementsPerWarp elements).
#pragma unroll
  for (int offset = 1; offset < 32; offset *= 2) {
    S += __shfl_down_sync(0xFFFFFFFF, S, offset);
  }

  float avg = S / elementsPerWarp;
  int opIndex = tid >> 5;

  // First thread in warp has the sum, write it in output.
  if (laneId == 0) {
    if (opIndex < outputSize) {
      if (prevLayerBias) avg += prevLayerBias[opIndex % C];
      output[opIndex] = avg;
    }
  }
}

template <typename T>
void globalAvgPool(int N, int C, T* output, const T* input,
                   const T* prevLayerBias) {
  const int kPlaneSize = 64;

  const bool fp16 = std::is_same<half, T>::value;
  if (fp16) {
    // For NHWC fp16, simply launch N blocks, each with C threads.
    globalAvgPool_kernel_NHWC_fp16<<<N, C>>>((half*)output, (half*)input,
                                             (half*)prevLayerBias,
                                             N * C * kPlaneSize, N * C);
  } else {
    // For NCHW layout (used with fp32),
    // each warp processes a full plane (64 elements), and writes a single
    // average N*C warps are launched.

    const int kTotalWarps = N * C;
    const int kWarpsPerBlock = 8;
    const int kBlockSize = kWarpsPerBlock * 32;

    int blocks = DivUp(kTotalWarps, kWarpsPerBlock);
    globalAvgPool_kernel<<<blocks, kBlockSize>>>((float*)output, (float*)input,
                                                 (float*)prevLayerBias,
                                                 N * C * kPlaneSize, N * C, C);
  }
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
void globalScale(int N, int C, T* output, const T* input, const T* scaleBias,
                 const T* prevLayerBias) {
  const bool fp16 = std::is_same<half, T>::value;

  // Each thread writes one output.
  const int kBlockSize = 256;
  const int kBlocks = DivUp(N * 8 * 8 * C, kBlockSize);

  if (fp16) {
    globalScale_kernel_fp16_nhwc<<<kBlocks, kBlockSize>>>(
        (half*)output, (half*)input, (half*)scaleBias, (half*)prevLayerBias,
        N * C * 8 * 8, C, 8 * 8 * C);
  } else {
    globalScale_kernel<<<kBlocks, kBlockSize>>>(
        (float*)output, (float*)input, (float*)scaleBias, (float*)prevLayerBias,
        N * C * 8 * 8, C);
  }
  ReportCUDAErrors(hipGetLastError());
}

template <typename T>
__global__ void policyMap_kernel(T* output, const T* input,
                                 const short* indices, int N, int inputSize,
                                 int usedSize, int outputSize) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  int n = tid / usedSize;
  int i = tid % usedSize;

  if (n >= N) return;

  int j = indices[i];

  if (j >= 0) {
    output[n * outputSize + j] = input[n * inputSize + i];
  }
}

template <typename T>
void PolicyMap(int N, T* output, const T* input, const short* indices,
               int inputSize, int usedSize, int outputSize) {
  // Each thread processes one input element
  // Only some of the threads (with valid mapping) write output
  const int kBlockSize = 256;
  const int kBlocks = DivUp(N * usedSize, kBlockSize);

  policyMap_kernel<T><<<kBlocks, kBlockSize>>>((T*)output, (T*)input,
                                               (short*)indices, N, inputSize,
                                               usedSize, outputSize);
  ReportCUDAErrors(hipGetLastError());
}

// Template instantiation.
template void copyTypeConverted<half, float>(half* op, float* ip, int N);
template void copyTypeConverted<float, half>(float* op, half* ip, int N);

template void batchNorm<float>(float* output, const float* input,
                               const float* skipInput, int N, int C, int H,
                               int W, float* means, float* var_multipliers,
                               bool relu);
template void batchNorm<half>(half* output, const half* input,
                              const half* skipInput, int N, int C, int H, int W,
                              float* means, float* var_multipliers, bool relu);

template void addVectors<float>(float* c, float* a, float* b, int size,
                                int asize, int bsize, bool relu, bool use_tanh,
                                bool use_sigmoid);
template void addVectors<half>(half* c, half* a, half* b, int size, int asize,
                               int bsize, bool relu, bool use_tanh,
                               bool use_sigmoid);

template void addBias_NCHW<float>(float* c, float* a, float* b, int N, int C,
                                  int H, int W);

template void addBias_NCHW<half>(half* c, half* a, half* b, int N, int C, int H,
                                 int W);

template void globalAvgPool<float>(int N, int C, float* output,
                                   const float* input,
                                   const float* prevLayerBias);
template void globalAvgPool<half>(int N, int C, half* output, const half* input,
                                  const half* prevLayerBias);

template void globalScale<float>(int N, int C, float* output,
                                 const float* input, const float* scaleBias,
                                 const float* prevLayerBias);
template void globalScale<half>(int N, int C, half* output, const half* input,
                                const half* scaleBias,
                                const half* prevLayerBias);

template void PolicyMap<float>(int N, float* output, const float* input,
                               const short* indices, int inputSize,
                               int usedSize, int outputSize);

template void PolicyMap<half>(int N, half* output, const half* input,
                              const short* indices, int inputSize, int usedSize,
                              int outputSize);

}  // namespace cudnn_backend
}  // namespace lczero
