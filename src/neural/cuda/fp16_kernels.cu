#include "hip/hip_runtime.h"
/*
  This file is part of Leela Chess Zero.
  Copyright (C) 2018 The LCZero Authors

  Leela Chess is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  Leela Chess is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with Leela Chess.  If not, see <http://www.gnu.org/licenses/>.

  Additional permission under GNU GPL version 3 section 7

  If you modify this Program, or any covered work, by linking or
  combining it with NVIDIA Corporation's libraries from the NVIDIA CUDA
  Toolkit and the NVIDIA CUDA Deep Neural Network library (or a
  modified version of those libraries), containing parts covered by the
  terms of the respective license agreement, the licensors of this
  Program grant you additional permission to convey the resulting work.
*/

#include "cuda_common.h"


namespace lczero {
namespace cudnn_backend {

/////////////////////////////////////////////////////////////////////////////
//          fp16-specific kernels used by certain layers                   //
/////////////////////////////////////////////////////////////////////////////



// SE layer implementation using single fused kernel.

// N blocks.
// C threads per block.
// 'HWC' input data processed by thread block.
// Each thread processes 8x8 elements.
// K is the no. of outputs of first fully connected layer (same as no. of inputs
// for second fully connected layer). 
// The kernel assumes K <= C.

#define readw1(row, col) (w1[(row)*K + (col)])
#define readw2(row, col) (w2[(row)*2 * C + (col)])

template <int C, int K>
__global__ void SE_Layer_NHWC(half* output, const half* skip, const half* input,
                              const half* w1, const half* b1, const half* w2,
                              const half* b2, const half *bPrev) {
  const int elementsPerThread = 64;  // 8x8 board

  int n = blockIdx.x;
  int c = threadIdx.x;

  __shared__ half sharedData[C];

  half2 localData[elementsPerThread];

  half S = 0;

  half bias = 0;
  if (bPrev) bias = bPrev[c];

  // 1. Global avg (1 avg per thread).
  #pragma unroll
  for (int i = 0; i < elementsPerThread; i++) {
    int localIndex = i * C + c;
    int inputIndex = n * C * elementsPerThread + localIndex;
    localData[i].x = input[inputIndex] + bias;
    localData[i].y = skip[inputIndex];
    S += localData[i].x;
  }

  half avg = S / (half)elementsPerThread;
  sharedData[c] = avg;

  __syncthreads();

  // 2. First fully connected layer.
  if (c < K) {
    S = 0;

    #pragma unroll
    for (int i = 0; i < C; i++) {
      S += sharedData[i] * readw1(i, c);
    }

    S += b1[c];

    // relu
    if (S < (half)0) S = 0;

    sharedData[c] = S;
  }
  __syncthreads();

  // 3. Second fully connected layer.
  S = 0;
  half B = 0;
  #pragma unroll
  for (int i = 0; i < K; i++) {
    half val = sharedData[i];
    S += val * readw2(i, c);
    B += val * readw2(i, c + C);
  }
  S += b2[c];
  B += b2[c + C];

  // Sigmoid (only on the scale part).
  S = (half)(1.0f / (1.0f + exp(-(float)(S))));

  // 4. Scale, and add skip connection, perform relu, and write to output.
  #pragma unroll
  for (int i = 0; i < elementsPerThread; i++) {
    int localIndex = i * C + c;
    int inputIndex = n * C * elementsPerThread + localIndex;
    half val = localData[i].y + localData[i].x * S + B;

    // Relu activation function.
    if (val < (half)0) val = 0;

    output[inputIndex] = val;
  }
}

bool Se_Fp16_NHWC(int N, int C, int numFc1Out, half* output, const half* skip,
                  const half* input, const half* w1, const half* b1,
                  const half* w2, const half* b2, const half* bPrev) {
  // TODO: Think of more elegant way to avoid this hardcoding :-/
  if (numFc1Out == 16) {
    if (C == 64) {
      SE_Layer_NHWC<64, 16>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else {
      // TODO: support other channel counts.
      throw Exception("channel count unsupported by SE layer");
    }
  } else if (numFc1Out == 32) {
    if (C == 64) {
      SE_Layer_NHWC<64, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else if (C == 128) {
      SE_Layer_NHWC<128, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else if (C == 192) {
      SE_Layer_NHWC<192, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else if (C == 256) {
      SE_Layer_NHWC<256, 32>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else {
      // TODO: support other channel counts.
      return false;
    }
  } else if (numFc1Out == 64) {
    if (C == 64) {
      SE_Layer_NHWC<64, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else if (C == 128) {
      SE_Layer_NHWC<128, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else if (C == 192) {
      SE_Layer_NHWC<192, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else if (C == 256) {
      SE_Layer_NHWC<256, 64>
          <<<N, C>>>(output, skip, input, w1, b1, w2, b2, bPrev);
    } else {
      // TODO: support other channel counts.
      return false;
    }
  } else {
    // TODO: support other sizes.
    return false;
  }
  ReportCUDAErrors(hipGetLastError());
  return true;
}

}   // namespace cudnn_backend
}   // namespace lczero
